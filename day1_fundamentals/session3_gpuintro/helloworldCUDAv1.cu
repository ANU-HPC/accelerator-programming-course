/*
 * Copyright 2019 Australian National University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either or express implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// this macro checks for errors in CUDA calls
#define Err(ans) \
  { gpucheck((ans), __FILE__, __LINE__); }
inline void gpucheck(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU Err: %s %s %d\n", hipGetErrorString(code),
            file, line);
    exit(code);
  }
}

__global__ void hello(char *res, int size) {
  char str[] = "Hello World!";
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    res[idx] = str[idx];
}

int main(void) {
  char *str_h, *str_d;
  const int size = 13;
  Err(hipHostMalloc(&str_h, size, hipHostMallocDefault)); // note we could just use a normal
                                     // malloc although this gives us
                                     // pinned memory
  Err(hipMalloc(&str_d, size));

  hello<<<1, 13>>>(str_d, size);
  Err(hipMemcpy(str_h, str_d, size, hipMemcpyDeviceToHost));

  printf("Result : %s\n", str_h);

  hipFree(str_h);
  hipFree(str_d);
}

