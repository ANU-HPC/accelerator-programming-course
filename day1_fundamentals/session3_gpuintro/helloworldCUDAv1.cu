#include <stdio.h>
#include <hip/hip_runtime.h>
 
// this macro checks for errors in cuda calls
#define Err(ans) { gpucheck((ans), __FILE__, __LINE__); }
inline void gpucheck(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU Err: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}
 
 
 
__global__ void hello(char *res, int size) {
    char str[] = "Hello World!";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) res[idx] = str[idx];
}

int main(void) {
    char *str_h, *str_d;
    const int size = 13;
    Err(hipHostMalloc(&str_h,size, hipHostMallocDefault));  // note we could just use a normal malloc although this gives us pinned memory 
    Err(hipMalloc(&str_d, size));  
 
    hello<<<1,13>>>(str_d, size);
    Err(hipMemcpy(str_h, str_d, size, hipMemcpyDeviceToHost));

    printf("Result : %s\n", str_h);
    
    hipFree(str_h); 
    hipFree(str_d);
}

