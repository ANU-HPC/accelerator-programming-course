#include <stdio.h>
#include <hip/hip_runtime.h>
 

// HelloworldCUDAv2 - uses the unified memory approach rather than explicitly copying the memory between 
//                    the device and the host. 

// this macro checks for errors in cuda calls
#define Err(ans) { gpucheck((ans), __FILE__, __LINE__); }
inline void gpucheck(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPU Err: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}
 

__global__ void hello(char *res) {
    char cstr[] = "Hello World!";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 13) res[idx] = cstr[idx];
}

int main(void) {
    char *str;
    const int size = 13;
    const int blocks = 1;
    Err(hipMallocManaged( &str, size));  
 
    hello<<<blocks,size>>>(str);
 
    Err(hipDeviceSynchronize());
    
    printf("Result : %s\n", str); 
    Err(hipFree(str));
}

